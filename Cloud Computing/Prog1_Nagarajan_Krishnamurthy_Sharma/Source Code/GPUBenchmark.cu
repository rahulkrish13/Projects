#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

int devCount, cudaCores, smCount, totalThreads;
long loop_count;
double f_avg, i_avg;
size_t global_mem;
struct timeval t;

// From Device Query Example code
inline int _ConvertSMVer2Cores(int major, int minor)
{
    // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
    typedef struct
    {
        int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] =
    {
        { 0x10,  8 }, // Tesla Generation (SM 1.0) G80 class
        { 0x11,  8 }, // Tesla Generation (SM 1.1) G8x class
        { 0x12,  8 }, // Tesla Generation (SM 1.2) G9x class
        { 0x13,  8 }, // Tesla Generation (SM 1.3) GT200 class
        { 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
        { 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
        { 0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
        { 0x32, 192}, // Kepler Generation (SM 3.2) GK10x class
        { 0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
        { 0x37, 192}, // Kepler Generation (SM 3.7) GK21x class
        { 0x50, 128}, // Maxwell Generation (SM 5.0) GM10x class
        {   -1, -1 }
    };
	    int index = 0;

    while (nGpuArchCoresPerSM[index].SM != -1)
    {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
        {
            return nGpuArchCoresPerSM[index].Cores;
        }

        index++;
    }

    // If we don't find the values, we default use the previous one to run properly
    printf("MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[index-1].Cores);
    return nGpuArchCoresPerSM[index-1].Cores;
}

// GPU FLOPS Benchmark

__global__ void FAdd(float *d_a, float *d_b, float *d_c, int totalThreads, long l_c)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < totalThreads)
	{
		for (int j = 0; j < l_c; ++j)
		{
			d_c[i] = d_c[i] + d_c[i];
                        d_b[i] = d_b[i] + d_b[i];
                        d_a[i] = d_a[i] + d_a[i];
                        d_c[i] = d_c[i] + d_c[i];
                        d_b[i] = d_b[i] + d_b[i];
                        d_a[i] = d_a[i] + d_a[i];
                        d_c[i] = d_c[i] + d_c[i];
                        d_b[i] = d_b[i] + d_b[i];
                        d_a[i] = d_a[i] + d_a[i];
                        d_b[i] = d_b[i] + d_b[i];
		}
	}
}

void FLOPSBenchmark()
{
	float *fa, *fb, *fc;
	float *d_fa, *d_fb, *d_fc;
	double fs_t, fe_t, ft_t;
	
	fa = (float *)malloc(totalThreads*sizeof(float));
	fb = (float *)malloc(totalThreads*sizeof(float));
	fc = (float *)malloc(totalThreads*sizeof(float));

	hipMalloc(&d_fa, totalThreads*sizeof(float));
	hipMalloc(&d_fb, totalThreads*sizeof(float));
	hipMalloc(&d_fc, totalThreads*sizeof(float));
	
	for(int i = 0; i < totalThreads; ++i)
	{
		fa[i] = 0.000001;
		fb[i] = 0.000001;
		fc[i] = 0.000001;
	}
	
	hipMemcpy(d_fa, fa, totalThreads*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_fb, fb, totalThreads*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_fc, fc, totalThreads*sizeof(float), hipMemcpyHostToDevice);
	
	gettimeofday(&t, NULL);
	fs_t = t.tv_sec+(t.tv_usec/1000000.0);
	
	FAdd<<< smCount, cudaCores >>>(d_fa, d_fb, d_fc, totalThreads, loop_count);
	
	hipDeviceSynchronize();
	
	gettimeofday(&t, NULL);
	fe_t = t.tv_sec+(t.tv_usec/1000000.0);
	ft_t = fe_t - fs_t;
	f_avg += (loop_count*smCount*cudaCores*totalThreads*10)/(ft_t*1000000000);

	free(fa);
	free(fb);
	free(fc);

	hipFree(d_fa);
	hipFree(d_fb);
	hipFree(d_fc);

}

// GPU IOPS Benchmark

__global__ void IAdd(int *d_a, int *d_b, int *d_c, int totalThreads, long l_c)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < totalThreads)
	{
		for (int j = 0; j < l_c; ++j)
		{
			d_c[i] = d_c[i] + d_c[i];
                        d_b[i] = d_b[i] + d_b[i];
                        d_a[i] = d_a[i] + d_a[i];
                        d_c[i] = d_c[i] + d_c[i];
                        d_b[i] = d_b[i] + d_b[i];
                        d_a[i] = d_a[i] + d_a[i];
                        d_c[i] = d_c[i] + d_c[i];
                        d_b[i] = d_b[i] + d_b[i];
                        d_a[i] = d_a[i] + d_a[i];
                        d_b[i] = d_b[i] + d_b[i];
		}
	}
}

void IOPSBenchmark()
{
	int *ia, *ib, *ic;
	int *d_ia, *d_ib, *d_ic;
	double is_t, ie_t, it_t;
	
	ia = (int *)malloc(totalThreads*sizeof(int));
	ib = (int *)malloc(totalThreads*sizeof(int));
	ic = (int *)malloc(totalThreads*sizeof(int));

	hipMalloc(&d_ia, totalThreads*sizeof(int));
	hipMalloc(&d_ib, totalThreads*sizeof(int));
	hipMalloc(&d_ic, totalThreads*sizeof(int));
	
	for(int i = 0; i < totalThreads; ++i)
	{
		ia[i] = 1;
		ib[i] = 1;
		ic[i] = 1;
	}
	
	hipMemcpy(d_ia, ia, totalThreads*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_ib, ib, totalThreads*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_ic, ic, totalThreads*sizeof(int), hipMemcpyHostToDevice);

	gettimeofday(&t, NULL);
        is_t = t.tv_sec + (t.tv_usec / 1000000.0);
	
	IAdd<<< smCount, cudaCores >>>(d_ia, d_ib, d_ic, totalThreads, loop_count);
	
	hipDeviceSynchronize();
	
	gettimeofday(&t, NULL);
	ie_t = t.tv_sec + ( t.tv_usec / 1000000.0);
	it_t = ie_t - is_t;
	i_avg += (loop_count*smCount*cudaCores*totalThreads*10)/(it_t*1000000000);
	
	free(ia);
	free(ib);
	free(ic);

	hipFree(d_ia);
	hipFree(d_ib);
	hipFree(d_ic);

}

// GPU Memory Benchmark

void MemBenchmark()
{
	char *ca, *cb;
	char *d_ca, *d_cb;
	long message_size = 1024*1024*1024;
	double ms_t, me_t, mt_t;
	
	ca = (char *)malloc(message_size*sizeof(char));
	cb = (char *)malloc(message_size*sizeof(char));
	
	for(int k = 0; k < message_size; ++k)
	{
		ca[k] = (char)((rand()%10)+97);
		cb[k] = '0';
	}
	
	// Sequential Read/Write
	for(int i=0; i<3; i++)
	{
		hipMalloc(&d_ca, message_size*sizeof(char));
		hipMalloc(&d_cb, message_size*sizeof(char));
		
		hipMemcpy(d_ca, ca, message_size*sizeof(char), hipMemcpyHostToDevice);
		
		if( i == 0)
		{
			gettimeofday(&t, NULL);
        		ms_t = t.tv_sec + (t.tv_usec / 1000000.0);
			for(int j = 0; j < message_size; ++j)
				hipMemcpy(&d_cb[j], &d_ca[j], 1, hipMemcpyDeviceToDevice);
			gettimeofday(&t, NULL);
        		me_t = t.tv_sec + ( t.tv_usec / 1000000.0);
        		mt_t = me_t - ms_t;
			printf("1 Byte/Sec Message Size: %f GBps\n", (message_size/(mt_t*1000000000)));
		}
		if( i == 1)
		{
			gettimeofday(&t, NULL);
                        ms_t = t.tv_sec + (t.tv_usec / 1000000.0);
			for(int j = 0; j < message_size; j += 1024)
				hipMemcpy(&d_cb[j], &d_ca[j], 1024, hipMemcpyDeviceToDevice);
			gettimeofday(&t, NULL);
                        me_t = t.tv_sec + ( t.tv_usec / 1000000.0);
                        mt_t = me_t - ms_t;
			printf("1 KByte/Sec Message Size: %f GBps\n", (message_size/(mt_t*1000000000)));
		}
		if( i == 2)
		{
			gettimeofday(&t, NULL);
                        ms_t = t.tv_sec + (t.tv_usec / 1000000.0);
			for(int j = 0; j < message_size ; j += (1024*1024))
				hipMemcpy(&d_cb[j], &d_ca[j], (1024*1024), hipMemcpyDeviceToDevice);
			gettimeofday(&t, NULL);
                        me_t = t.tv_sec + ( t.tv_usec / 1000000.0);
                        mt_t = me_t - ms_t;
			printf("1 MByte/Sec Message Size: %f GBps\n", (message_size/(mt_t*1000000000)));
		}
		hipFree(d_ca);
		hipFree(d_cb);
		sleep(1);
	}
	free(ca);
	free(cb);
}

int main(int argc, char *argv[])
{

	f_avg = 0;
	i_avg = 0;
	loop_count = atoi(argv[1]); // loop count
	hipGetDeviceCount(&devCount);
	printf("\nCUDA Device Query...\n");
	
	// Check CUDA Device Present
	if( devCount == 0 )
	{
		printf("No CUDA device found, Ending...");
	}
	else
	{
		//CUDA Device Count
		printf("There are %d CUDA devices.\n", devCount);

		// Iterate through devices
		for (int i = 0; i < devCount; ++i)
		{
			hipSetDevice(i);
			// Get device properties
			printf("\nCUDA Device #%d\n\n", (i+1));
			hipDeviceProp_t devProp;
			hipGetDeviceProperties(&devProp, i);
			smCount = devProp.multiProcessorCount;
			cudaCores = _ConvertSMVer2Cores(devProp.major, devProp.minor);
			totalThreads = smCount * cudaCores;
			global_mem = devProp.totalGlobalMem;
			printf("CUDA Capability <Major.Minor> version number:     %d.%d\n", devProp.major, devProp.minor);
			printf("%d Multiprocessors, %d CUDA Cores/MP:     %d CUDA Cores\n", smCount, cudaCores, totalThreads);
			printf("Total amount of Global memory:               %lu GBytes\n", global_mem/(1024*1024*1024));
			printf("GPU Clock Rate:	%.0f MHz (%0.2f GHz)\n", devProp.clockRate * 1e-3f, devProp.clockRate * 1e-6f);
			
			//FLOPS Benchmark
			printf("\nStarting FLOPs Benchmark...\n");
			for( int h = 1; h <= 5; h++)
			{
				FLOPSBenchmark();
			}
			printf("Number of FLOPs: %lf G-FLOPs\n", (f_avg/5));
			sleep(1);

			//IOPS Benchmark
			printf("\nStarting IOPs Benchmark...\n");
			for( int h = 1; h <= 5; h++)
			{
				IOPSBenchmark();
			}
			printf("Number of IOPs: %lf G-IOPs\n", (i_avg/5));
			sleep(1);

			//Memory Benchmark
			printf("\nStarting Memory Benchmark...\n");
			MemBenchmark();
			
			printf("\nEnding...\n");
			
		}
	}

	return 0;
}
